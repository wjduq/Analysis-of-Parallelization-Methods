#include <hip/hip_runtime.h>
#include <iostream>
#include <omp.h>
#include <ctime>
#include <time.h>

using namespace std;

const int N = 1 << 20;
#define MAX 4

__global__
void count3s_array(int *a, int* count)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(a[i] == 3)
  {
    count[i] = 1;
  }else{
    count[i] = 0;
  }
}


void getRandomArray(int *data_ptr, int data_len)
{
  for(int i = 0; i < data_len; i++)
  {
    data_ptr[i] = rand() % MAX;
  }
}


int main()
{

  srand(time(0));
  int len = 32768; //change this in order to change the size of the array
  int *count_array = (int *)malloc(len*sizeof(int));
  int *count_array_null = (int *)malloc(len*sizeof(int));
  int *test_array = (int *)malloc(len*sizeof(int));

  getRandomArray(test_array, len);

 //for checking correctness. Prints out array
 /*
  for(int i = 0; i<len; i++)
  {
    cout << test_array[i];
  }cout << endl;*/


  //create streams
  const int num_streams = 8;

  hipStream_t streams[num_streams];
  float *data[num_streams];

  //allocate gpu memory
  int *d_test_array;
  int *d_count_array;
  int *d_count_array_null;
  hipMalloc(&d_test_array, len*sizeof(int));
  hipMalloc(&d_count_array, len*sizeof(int));
  hipMalloc(&d_count_array_null, len*sizeof(int));

  //copy host to device
  hipMemcpy(d_test_array, test_array, len*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_count_array, count_array, len*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_count_array_null, count_array_null, len*sizeof(int), hipMemcpyHostToDevice);

  struct timespec begin_stream, end_stream;
  clock_gettime(CLOCK_REALTIME, &begin_stream);
  int threadsPerBlock;

  // must be power of two that is smaller than the length of the array. Ideally, you want
  // the threads per block to be as large as possible so long as they are under 1024
  if (len < 1024){
    threadsPerBlock = len;
  }
  else{
    threadsPerBlock = 1024;
  }

  //int threadsPerBlock = 8; // must be power of two that is smaller than the length of the array
  int numBlocks = len/threadsPerBlock;
  for (int i = 0; i < num_streams; ++i){

    hipStreamCreate(&streams[i]); //Create an asynchronous stream

    hipMalloc(&data[i], N * sizeof(float));

    // Launch one worker kernal per streams
    count3s_array<<<numBlocks, threadsPerBlock, 0, streams[i]>>>(d_test_array, d_count_array);
  }

  hipDeviceSynchronize(); //Blocks until the device has completed all preceding requested tasks

  for (int i = 0; i < num_streams; ++i){
    hipStreamDestroy(streams[i]); //Destroys and cleans up an asynchronous stream
  }

  //TIME USING NULL STREAM------------------------------------------------------
  struct timespec begin_null, end_null;
  clock_gettime(CLOCK_REALTIME, &begin_null);
  // No Streams
  count3s_array<<<numBlocks, threadsPerBlock>>>(d_test_array, d_count_array_null);
  hipDeviceSynchronize();

  //copy device to host
  hipMemcpy(count_array, d_count_array, len*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(count_array_null, d_count_array_null, len*sizeof(int), hipMemcpyDeviceToHost);

  int totalnull = 0;

  // Sum the count array from null stream
  #pragma omp parallel for reduction(+ : total)
  for (int i = 0; i < len; ++i){
    totalnull += count_array_null[i];
  }

  clock_gettime(CLOCK_REALTIME, &end_null); // end null timing

  int totalstream = 0;

  #pragma omp parallel for reduction(+ : total)
  for (int i = 0; i < len; ++i){
    totalstream += count_array[i];
  }
  clock_gettime(CLOCK_REALTIME, &end_stream);

  // Streams calc
  long seconds_stream = end_stream.tv_sec - begin_stream.tv_sec;
  long nanos_stream = end_stream.tv_nsec - begin_stream.tv_nsec;
  double elapsed_stream = seconds_stream*1000 + nanos_stream*1e-6; //convert to ms

  // Null calc
  long seconds_null = end_null.tv_sec - begin_null.tv_sec;
  long nanos_null = end_null.tv_nsec - begin_null.tv_nsec;
  double elapsed_null = seconds_null*1000 + nanos_null*1e-6; //convert to ms

  cout << "Timing with Null: "<< elapsed_null << " ms"<<endl;
  cout << "GPU NULL Count is: "<< totalnull << endl;

  //We subtract elapsed_null in order to take account for the time that executing
  //the kernel using only the null stream takes and so that doesn't impact the overall
  //time of our kernel with streams
  cout << "Timing with Stream: "<< elapsed_stream-elapsed_null << " ms"<<endl;
  cout<<"GPU Stream Count is: "<< totalstream << endl;

  //free gpu memory
  hipFree(d_count_array);
  hipFree(d_test_array);
  free(test_array);
  free(count_array);


  return 0;
}
